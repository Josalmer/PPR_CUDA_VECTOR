
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string.h>
#include <time.h>
#include <math.h>

using namespace std;

int index(int i) { return i + 2; }
// Blocksize
#define BLOCKSIZE 1024
// Number of mesh points
int n = 60000;

//*************************************************
// Swap two pointers to float
// ************************************************
void swap_pointers(float **a, float **b) {
  float *tmp = *a;
  *a = *b;
  *b = tmp;
}

//*************************************************
// GLOBAL MEMORY  VERSION OF THE FD UPDATE
// ************************************************
__global__ void vectorNS(float *d_phi, float *d_phi_new, float cu, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x + 2;

  // Inner point update
  if (i < n + 3) {
    d_phi_new[i] = (d_phi[i - 2] * d_phi[i - 2] + 2 * d_phi[i - 1] * d_phi[i - 1] + d_phi[i] * d_phi[i] - 3 * d_phi[i + 1] * d_phi[i +1] + 5 * d_phi[i + 2] * d_phi[i + 2]) / 24;
  }

  // Boundary Conditions
  if (i == 2) {
    d_phi_new[0] = 0;
    d_phi_new[1] = 0;
  }
  if (i == n + 2) {
    d_phi_new[n + 3] = 0;
    d_phi_new[n + 4] = 0;
  }
}

//*************************************************
// TILING VERSION  (USES SHARED MEMORY) OF THE FD UPDATE
// ************************************************
__global__ void FD_kernel2(float *d_phi, float *d_phi_new, float cu, int n) {
  int li = threadIdx.x + 1;                           //local index in shared memory vector
  int gi = blockDim.x * blockIdx.x + threadIdx.x + 1; // global memory index
  int lstart = 0;
  int lend = BLOCKSIZE + 1;
  __shared__ float s_phi[BLOCKSIZE + 2]; //shared mem. vector
  float result;

  // Load Tile in shared memory
  if (gi < n + 2) {
    s_phi[li] = d_phi[gi];
  }

  if (threadIdx.x == 0) { // First Thread (in the current block)
    s_phi[lstart] = d_phi[gi - 1];
  }

  if (threadIdx.x == BLOCKSIZE - 1) { // Last Thread
    if (gi >= n + 1) {                // Last Block
      s_phi[(n + 2) % BLOCKSIZE] = d_phi[n + 2];
    } else {
      s_phi[lend] = d_phi[gi + 1];
    }
  }
  __syncthreads();

  if (gi < n + 2) {
    // Lax-Friedrichs Update
    result = 0.5 * ((s_phi[li + 1] + s_phi[li - 1]) - cu * (s_phi[li + 1] - s_phi[li - 1]));
    d_phi_new[gi] = result;
  }

  // Boundary Conditions
  if (gi == 1) {
    d_phi_new[0] = d_phi_new[1];
  }
  if (gi == n + 1) {
    d_phi_new[n + 2] = d_phi_new[n + 1];
  }
}

//******************************
//**** MAIN FUNCTION ***********

int main(int argc, char *argv[]) {

  //******************************
  //Get GPU information
  int devID;
  hipDeviceProp_t props;
  hipError_t err;
  err = hipGetDevice(&devID);
  if (err != hipSuccess) {
    cout << "ERRORRR" << endl;
  }
  hipGetDeviceProperties(&props, devID);
  printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);

  cout << "Introduce number of points (1000-200000)" << endl;
  cin >> n;

  // Domain size (periodic)
  float l = 10.0;
  // Grid
  float dx = l / n;
  // Advecting velocity
  float u = 1.0;

  //Timestep size
  float dt = 0.8 * u * dx;
  float tend = 2.5;
  // Courant number
  float cu = u * dt / dx;

  //Number of steps to take
  int nsteps = (int)ceil(tend / dt);

  cout << "dx=" << dx << "...  dt= " << dt << "...Courant= " << cu << endl;
  cout << endl;
  cout << "Number of time steps=" << nsteps << endl;

  //Mesh Definition    blockDim.x*blockIdx.x
  float *phi = new float[n + 5];
  float *phi_new = new float[n + 5];
  float *phi_GPU = new float[n + 5];
  float xx[n + 2];

  for (int i = 0; i <= n; i++) {
    xx[i] = -5.0 + i * dx;
  }

  // Initial values for phi--> Gaussian
  for (int i = 0; i <= n; i++) {
    // Gaussian
    phi[index(i)] = (1.0 / (2.0 * M_PI * 0.16)) * exp(-0.5 * (pow((xx[i] - 0.5), 2) / 0.01));
  }

  //**************************
  // GPU phase
  //**************************
  int size = (n + 3) * sizeof(float);

  // Allocation in device mem. for d_phi
  float *d_phi = NULL;
  err = hipMalloc((void **)&d_phi, size);
  if (err != hipSuccess) {
    cout << "ALLOCATION ERROR" << endl;
  }
  // Allocation in device mem. for d_phi_new
  float *d_phi_new = NULL;
  err = hipMalloc((void **)&d_phi_new, size);
  if (err != hipSuccess) {
    cout << "ALLOCATION ERROR" << endl;
  }

  // Take initial time
  cout << "Start GPU" << endl;
  double t1 = clock();

  // Impose Boundary Conditions
  phi[index(-2)] = 0;
  phi[index(-1)] = 0;
  phi[index(n + 1)] = 0;
  phi[index(n + 2)] = 0;

  // Copy phi values to device memory
  err = hipMemcpy(d_phi, phi, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    cout << "GPU COPY ERROR" << endl;
  }
  // *******************
  // Time Step Iteration
  // *******************
  for (int k = 0; k < nsteps; k++) {
    int blocksPerGrid = (int)ceil((float)(n + 2) / BLOCKSIZE);

    // ********* Kernel Launch ************************************
    vectorNS<<<blocksPerGrid, BLOCKSIZE>>>(d_phi, d_phi_new, cu, n);
    // ************************************************************

    err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel! %d \n", err);
      exit(EXIT_FAILURE);
    }
    swap_pointers(&d_phi, &d_phi_new);
  }

  hipMemcpy(phi_GPU, d_phi, size, hipMemcpyDeviceToHost);

  double Tgpu = clock();
  Tgpu = (Tgpu - t1) / CLOCKS_PER_SEC;
  cout << "End GPU" << endl;

  //**************************
  // CPU phase
  //**************************

  cout << "Start CPU" << endl;
  double t1cpu = clock();    
  
  // Impose Boundary Conditions
  phi[index(-2)] = 0;
  phi[index(-1)] = 0;
  phi[index(n + 1)] = 0;
  phi[index(n + 2)] = 0;
  for (int k = 0; k < nsteps; k++) {
    for (int i = 0; i <= n; i++) {
      //Lax-Friedrichs
      phi_new[index(i)] = (phi[index(i - 2)] * phi[index(i - 2)] + 2 * phi[index(i - 1)] *phi[index(i - 1)] + phi[index(i)] * phi[index(i)] - 3 * phi[index(i + 1)] * phi[index(i + 1)] + 5 * phi[index(i + 2)] * phi[index(i + 2)]) / 24;
    }
    swap_pointers(&phi, &phi_new);
  }

  double Tcpu = clock();
  Tcpu = (Tcpu - t1cpu) / CLOCKS_PER_SEC;
  cout << "End CPU" << endl;

  cout << endl;
  cout << "GPU Time= " << Tgpu << endl << endl;
  cout << "CPU Time= " << Tcpu << endl << endl;

  //**************************
  // CPU-GPU comparison and error checking
  //**************************

  int passed = 1;
  int i = 0;
  while (passed && i < n) {
    double diff = fabs((double)phi_GPU[index(i)] - (double)phi[index(i)]);
    if (diff > 1.0e-5) {
      passed = 0;
      cout << "DIFF= " << diff << endl;
    }
    i++;
  }

  if (passed) {
    cout << "PASSED TEST !!!" << endl;
  }
  else {
    cout << "ERROR IN TEST !!!" << endl;
  }

  cout << endl;
  cout << "Speedup (T_CPU/T_GPU)= " << Tcpu / Tgpu << endl;

  return 0;
}

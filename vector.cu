
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>

using namespace std;
// Blocksize
#define BLOCKSIZE 1024

//*************************************************
// GLOBAL MEMORY  VERSION OF THE ALGORITHM
// ************************************************
__global__ void vectorNS(float *in, float *out, int n) {

  int i = threadIdx.x + blockDim.x * blockIdx.x + 2;
  int iB = i - 2;
  if (iB < n) {
    float Aim2 = in[i - 2];
    float Aim1 = in[i - 1];
    float Ai = in[i];
    float Aip1 = in[i + 1];
    float Aip2 = in[i + 2];
    out[iB] = (pow(Aim2, 2) + 2.0 * pow(Aim1, 2) + pow(Ai, 2) - 3.0 * pow(Aip1, 2) + 5.0 * pow(Aip2, 2)) / 24.0;
  }
}

//*************************************************
// TILING VERSION  (USES SHARED MEMORY) OF THE ALGORITHM
// ************************************************
__global__ void vectorS(float *in, float *out, int n) {
  int li = threadIdx.x + 2;                           //local index in shared memory vector
  int gi = blockDim.x * blockIdx.x + threadIdx.x + 2; // global memory index
  int lstart = 0;
  int lend = BLOCKSIZE + 2;
  __shared__ float s_phi[BLOCKSIZE + 4]; //shared mem. vector

  // Load Tile in shared memory
  if (gi < n + 3) {
    s_phi[li] = in[gi];
  }

  if (threadIdx.x == 0) { // First Thread (in the current block)
    s_phi[lstart] = in[gi - 2];
    s_phi[lstart + 1] = in[gi - 1];
  }

  if (threadIdx.x == BLOCKSIZE - 1) { // Last Thread
    if (gi >= n + 1) {                // Last Block
      s_phi[(n + 2) % BLOCKSIZE] = in[n + 2];
    } else {
      s_phi[lend - 1] = in[gi + 1];
      s_phi[lend] = in[gi + 2];
    }
  }
  __syncthreads();

  if (gi < n + 2) {
    out[gi] = (s_phi[li - 2] * s_phi[li - 2] + 2 * s_phi[li - 1] * s_phi[li - 1] + s_phi[li] * s_phi[li] - 3 * s_phi[li + 1] * s_phi[li +1] + 5 * s_phi[li + 2] * s_phi[li + 2]) / 24;
  }
}

//**************************************************************************
// FIND MAX IN VECTOR
__global__ void reduceMax(float * V_in, float * V_out, const int N) {
	extern __shared__ float sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = ((i < N) ? V_in[i] : -1);
	__syncthreads();

	for(int s = blockDim.x/2; s > 0; s >>= 1) {
	  if (tid < s) {
      if(sdata[tid] < sdata[tid + s]) {
        sdata[tid] = sdata[tid + s];
      }
	  }
	  __syncthreads();
	}
	if (tid == 0) {
		V_out[blockIdx.x] = sdata[0];
	}
}

//**************************************************************************
int main(int argc, char *argv[]) {
  //******************************
  //Get GPU information
  int devID;
  hipDeviceProp_t props;
  hipError_t err;
  err = hipGetDevice(&devID);
  if (err != hipSuccess) {
    cout << "ERRORRR" << endl;
  }
  hipGetDeviceProperties(&props, devID);
  printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);

  int N;
  if (argc != 2) {
    cout << "Uso: transformacion Num_elementos  " << endl;
    return (0);
  }
  else {
    N = atoi(argv[1]);
  }

  //* pointers to host memory */
  float *A, *B;

  //* Allocate arrays a, b and c on host*/
  A = new float[N + 4];
  B = new float[N];
  float mx; // maximum of B

  //* Initialize array A */
  for (int i = 2; i < N + 2; i++)
    A[i] = (float)(1 - (i % 100) * 0.001);

  // Impose Boundary Conditions
  A[0] = 0.0;
  A[1] = 0.0;
  A[N + 2] = 0.0;
  A[N + 3] = 0.0;

  //**************************
  // GPU phase
  //**************************
  float *B_GPU = new float[N];

  int Nsize = N * sizeof(float);
  int NsizeWithBound = (N + 4) * sizeof(float);
  // Allocation in device mem
  float *A_GPU = NULL;
  err = hipMalloc((void **)&A_GPU, NsizeWithBound);
  if (err != hipSuccess) {
    cout << "ALLOCATION ERROR" << endl;
  }
  float *out = NULL;
  err = hipMalloc((void **)&out, Nsize);
  if (err != hipSuccess) {
    cout << "ALLOCATION ERROR" << endl;
  }

  // Take initial time
  cout << "Start GPU" << endl;
  double gt1 = clock();

  // Copy A values to device memory
  err = hipMemcpy(A_GPU, A, NsizeWithBound, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    cout << "GPU COPY ERROR" << endl;
  }

  int blocksPerGrid = (int)ceil((float)(N) / BLOCKSIZE);

  cout << endl;
  // ********* Kernel Launch ************************************
  vectorNS<<<blocksPerGrid, BLOCKSIZE>>>(A_GPU, out, N);
  // ************************************************************

  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch kernel! %d \n", err);
    exit(EXIT_FAILURE);
  }

  hipMemcpy(B_GPU, out, Nsize, hipMemcpyDeviceToHost);

  double Tgpu = clock();
  Tgpu = (Tgpu - gt1) / CLOCKS_PER_SEC;
  cout << "End GPU" << endl;

  //**************************
  // CPU phase
  //**************************
  cout << "Start CPU" << endl;
  // Time measurement
  double ct1 = clock();

  float Ai, Aim1, Aim2, Aip1, Aip2;
  // Compute B[i] and mx
  for (int i = 2; i < N + 2; i++) {
    const int iB = i - 2;
    Aim2 = A[i - 2];
    Aim1 = A[i - 1];
    Ai = A[i];
    Aip1 = A[i + 1];
    Aip2 = A[i + 2];
    B[iB] = (pow(Aim2, 2) + 2.0 * pow(Aim1, 2) + pow(Ai, 2) - 3.0 * pow(Aip1, 2) + 5.0 * pow(Aip2, 2)) / 24.0;
    mx = (iB == 0) ? B[0] : max(B[iB], mx);
  }

  double Tcpu = clock();
  Tcpu = (Tcpu - ct1) / CLOCKS_PER_SEC;
  cout << "End CPU" << endl;

  //**************************
  // CPU-GPU comparison and error checking
  //**************************

  int passed = 1;
  int i = 0;
  while (passed && i < N) {
    float diff = fabs(B[i] - B_GPU[i]);
    if (diff > 0) {
      passed = 0;
      cout << endl << i << endl;
      cout << "DIFF= " << diff << endl;
    }
    i++;
  }

  if (passed) {
    cout << "PASSED TEST !!!" << endl;
  }
  else {
    cout << "ERROR IN TEST !!!" << endl;
  }

	// c_d Maximum computation on GPU
	dim3 threadsPerBlock(BLOCKSIZE);
	dim3 numBlocks( ceil ((float)(N)/threadsPerBlock.x));

	// Maximum vector on CPU
	float * vmax;
	vmax = (float*) malloc(numBlocks.x*sizeof(float));

	// Maximum vector  to be computed on GPU
	float *vmax_d; 
	hipMalloc ((void **) &vmax_d, sizeof(float)*numBlocks.x);

	float smemSize = threadsPerBlock.x*sizeof(float);

	// Kernel launch to compute Minimum Vector
	reduceMax<<<numBlocks, threadsPerBlock, smemSize>>>(out,vmax_d, N);


	/* Copy data from device memory to host memory */
	hipMemcpy(vmax, vmax_d, numBlocks.x*sizeof(float),hipMemcpyDeviceToHost);

	// Perform final reduction in CPU
	float max_gpu = -1;
	for (int i=0; i<numBlocks.x; i++) {
		max_gpu =max(max_gpu,vmax[i]);
	}

  if (N < 16) {
    for (int i = 0; i < N; i++) {
      cout << "CPU[" << i << "] = " << B[i] << ", GPU[" << i << "] = " << B_GPU[i] << endl;
    }
  }
  cout << "................................." << endl;
  cout << "................................." << endl
       << "El valor máximo en B es (CPU):  " << mx << endl;
  cout << "................................." << endl
       << "El valor máximo en B es (GPU):  " << max_gpu << endl;
  cout << endl
       << "Tiempo gastado CPU= " << Tcpu << endl
       << endl;
  cout << endl
       << "Tiempo gastado GPU= " << Tgpu << endl
       << endl;
  cout << endl
       << "Speedup GPU= " << Tcpu / Tgpu << endl
       << endl;

  //* Free the memory */
  delete (A);
  delete (B);
  hipFree(A_GPU);
  hipFree(B_GPU);
  hipFree(out);
}
